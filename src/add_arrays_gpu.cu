#include "hip/hip_runtime.h"
#include "add_arrays.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void add_kernel(const float* a, const float* b, float* result, size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        // result[idx] = a[idx] + b[idx];
        result[idx] = add_op(a[idx], b[idx]);
    }
}

void add_arrays(const float* a, const float* b, float* result, size_t size) {
    printf("I am running on a GPU\n");
    float *d_a, *d_b, *d_result;
    size_t bytes = size * sizeof(float);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_result, bytes);

    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(d_a, d_b, d_result, size);

    hipMemcpy(result, d_result, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
